#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <vector>
#include <iostream>
#include <vector>
#include <algorithm>
#include <string>
#include <time.h>
#include <stdexcept>
#include <cmath>

using namespace std;

#define GAME_SIZE 3
#define MAX_CHILD 4

vector<string> states_checked;
//__device__ volatile __managed__ bool  isFinished = false;
__device__ bool  isFinished = false;
__managed__ int  total_threads = 640;
int blocks = 10;

hipError_t bfsWithCuda(__int8* memory, struct node* nodes, int* heigths);

struct state
{
    int game_matrix[GAME_SIZE][GAME_SIZE];
    int zero_index[2];

    state() {}

    state(int actual_matrix[GAME_SIZE][GAME_SIZE]) {
        for (int i = 0; i < GAME_SIZE; i++)
            for (int j = 0; j < GAME_SIZE; j++) {
                if (actual_matrix[i][j] == 0) {
                    zero_index[0] = i;
                    zero_index[1] = j;
                }
                game_matrix[i][j] = actual_matrix[i][j];
            }
    }

};

bool validateState(struct state actual_state) {
    for (int i = 0; i < GAME_SIZE; i++)
        for (int j = 0; j < GAME_SIZE; j++)
            if (actual_state.game_matrix[i][j] != j + (GAME_SIZE * i)) return false;
    return true;
}

string generateStateHash(struct state actual_state) {
    string hash_id = "";
    for (int i = 0; i < GAME_SIZE; i++)
        for (int j = 0; j < GAME_SIZE; j++)
            hash_id += to_string(actual_state.game_matrix[i][j]);

    return hash_id;
}

bool verifyStateChecked(struct state actual_state) {
    string hash_id = generateStateHash(actual_state);
    return find(states_checked.begin(), states_checked.end(), hash_id) != states_checked.end();
}

struct node
{
    vector<int> secuence;
    struct state node_state;

    node() {}

    node(struct state problem_state) {
        node_state = problem_state;
    }
};

__device__ void dev_printState(__int8 matrix[3][3]) {
    printf("Hilo numero: %d\n%d %d %d \n%d %d %d \n%d %d %d \n\n",
        threadIdx.x + blockIdx.x * blockDim.x,
        matrix[0][0],
        matrix[0][1],
        matrix[0][2],
        matrix[1][0],
        matrix[1][1],
        matrix[1][2],
        matrix[2][0],
        matrix[2][1],
        matrix[2][2]);
}

__global__ void addKernel(__int8* memory, struct node* nodes, int* heights)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int const registryMemorySpace = 6000; //4000 para 18 movimientos // Se puede hacer variable de acuerdo con el numero de hilos por bloque (?)
    int const stackSize = (registryMemorySpace*2)/3;
    int const maxMemorySize = registryMemorySpace/3;
    
    __int8 stack[stackSize][3][3]; //Max 1820
    __int8 nodeMovements[stackSize];
    __int8 re_memory[maxMemorySize][3][3];

    int actualIndex = 0;
    int size = 1;
    int memorySize = 1;

    int nodesChecked = 1; //DEBUG
    int nodesCreated = 1; //DEBUG
    int resetTimes = 0;   //DEBUG
    int isUseful = 0;     //DEBUG
    int memoryNeeded = 0; //DEBUG

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            stack[actualIndex][i][j] = nodes[idx].node_state.game_matrix[i][j];
            re_memory[0][i][j] = memory[(j + (i * 3)) + (idx * 9)];
        }
    }
    nodeMovements[actualIndex] = heights[idx];

    while (size < registryMemorySpace && !isFinished) {

        bool ansFound = true;
        int i_0 = -1;
        int j_0 = -1;

        for (int i = 0; i < 3; i++) {
            //if (!ansFound && i_0 != -1 && j_0 != -1) break;
            for (int j = 0; j < 3; j++) {
                if (stack[actualIndex][i][j] != j + (3 * i)) ansFound = false;
                if (stack[actualIndex][i][j] == 0) {
                    i_0 = i;
                    j_0 = j;
                }
            }
        }

        if (ansFound) {
            printf("\nHILO %d: Encuentra una solucion con %d movimientos y con un size de pila de %d\n\n", idx, nodeMovements[actualIndex], size);
            dev_printState(stack[actualIndex]);
            dev_printState(stack[0]);
            isFinished = true;
        }else {
            for (int i = 1; i <= 4; i++) {
                
                int provIndex = (actualIndex + size) % registryMemorySpace; //Se consideran los límites de la memoria de la stack
                bool isValid = false;

                if (provIndex==0) resetTimes++; //DEBUG

                switch (i) {
                    case 1:
                        isValid = i_0 != 0;
                        break;
                    case 2:
                        isValid = j_0 != 2;
                        break;
                    case 3:
                        isValid = i_0 != 2;
                        break;
                    case 4:
                        isValid = j_0 != 0;
                        break;
                }

                if (isValid && size<= registryMemorySpace) {

                    for (int j = 0; j < 3; j++) {
                        for (int k = 0; k < 3; k++) {
                            stack[provIndex][j][k] = stack[actualIndex][j][k];
                        }
                    }
                    nodeMovements[provIndex] = nodeMovements[actualIndex]+1;

                    switch (i) {
                        case 1:
                            stack[provIndex][i_0][j_0] = stack[provIndex][i_0 - 1][j_0];
                            stack[provIndex][i_0 - 1][j_0] = 0;
                            break;
                        case 2:
                            stack[provIndex][i_0][j_0] = stack[provIndex][i_0][j_0 + 1];
                            stack[provIndex][i_0][j_0 + 1] = 0;
                            break;
                        case 3:
                            stack[provIndex][i_0][j_0] = stack[provIndex][i_0 + 1][j_0];
                            stack[provIndex][i_0 + 1][j_0] = 0;
                            break;
                        case 4:
                            stack[provIndex][i_0][j_0] = stack[provIndex][i_0][j_0 - 1];
                            stack[provIndex][i_0][j_0 - 1] = 0;
                            break;
                    }

                    bool found = false;
                    for (int j = 0; j < memorySize; j++) {
                        if (!found) {
                            found = true;
                            for (int k = 0; k < 3; k++) {
                                for (int m = 0; m < 3; m++) {
                                    if(found) found = re_memory[j][k][m] == stack[provIndex][k][m];
                                }
                            }
                        }
                    }

                    if (!found) {
                        size++;
                        nodesCreated++; //DEBUG
                    }
                    else  isUseful++;     //DEBUG
                }
            }

            if (memorySize < maxMemorySize) {
                for (int i = 0; i < 3; i++) {
                    for (int j = 0; j < 3; j++) {
                        re_memory[memorySize][i][j] = stack[actualIndex][i][j];
                    }
                }
                memorySize++;
            }else memoryNeeded++; //DEBUG

            size--;
            actualIndex = (actualIndex + 1) % registryMemorySpace;
            nodesChecked++; //DEBUG
        }
    }

    if (!isFinished && idx == 1) {
        printf("--- NO SE ENCUENTRA UNA SOLUCION, SE NECESITA MAS MEMORIA ---\n\n");
    }

    //printf("El hilo %d ha evaluado %d nodos, en ese numero se han generado en total %d nodos.\nSe ha sobreescrito la posicion 0 de la cola %d veces.\nLa memoria compartida fue util %d veces, pero se necesito mas memoria %d veces.\n\Por otro lado, la memoria se utilizo %d veces.\n\n", idx, nodesChecked, nodesCreated, resetTimes, isUseful, memoryNeeded, memorySize);
    //DEBUG

    return;
}

struct state generateState(struct state actual_state, int movement) {
    int i_0 = actual_state.zero_index[0];
    int j_0 = actual_state.zero_index[1];

    switch (movement) {
    case 1:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0 - 1][j_0];
        actual_state.game_matrix[i_0 - 1][j_0] = 0;

        actual_state.zero_index[0] = i_0 - 1;
        break;
    case 2:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0][j_0 + 1];
        actual_state.game_matrix[i_0][j_0 + 1] = 0;

        actual_state.zero_index[1] = j_0 + 1;
        break;
    case 3:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0 + 1][j_0];
        actual_state.game_matrix[i_0 + 1][j_0] = 0;

        actual_state.zero_index[0] = i_0 + 1;
        break;
    case 4:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0][j_0 - 1];
        actual_state.game_matrix[i_0][j_0 - 1] = 0;

        actual_state.zero_index[1] = j_0 - 1;
        break;
    }

    return actual_state;
}

bool verifyMovement(struct state actual_state, int movement) {
    int i_0 = actual_state.zero_index[0];
    int j_0 = actual_state.zero_index[1];

    switch (movement) {
    case 1:
        return i_0 != 0;
        break;
    case 2:
        return j_0 != GAME_SIZE - 1;
        break;
    case 3:
        return i_0 != GAME_SIZE - 1;
        break;
    case 4:
        return j_0 != 0;
        break;
    default:
        return false;
        break;
    }
}

vector<node> createChildren(struct node actual_node) {

    struct state actual_state = actual_node.node_state;
    vector<node> children;

    for (int i = 1; i <= MAX_CHILD; i++) {
        if (verifyMovement(actual_state, i)) {
            struct state new_state = generateState(actual_state, i);
            if (!verifyStateChecked(new_state)) {
                states_checked.push_back(generateStateHash(new_state));

                struct node new_node(new_state);
                new_node.secuence = actual_node.secuence;
                new_node.secuence.push_back(i);

                children.push_back(new_node);
            }
        }
    }

    return children;
}

void print_state(struct state actual_state) {
    for (int i = 0; i < GAME_SIZE; i++) {
        for (int j = 0; j < GAME_SIZE; j++)
            printf("%d ", actual_state.game_matrix[i][j]);
        printf("\n");
    }
    printf("\n");
}

vector<node> distributeThreads(int num_threads, vector<node> nodes) {

    struct node initial_node = nodes.front();
    states_checked.push_back(generateStateHash(initial_node.node_state));

    while (nodes.size() < num_threads) {
        struct node actual_node = nodes.front();

        if (validateState(actual_node.node_state)) return { actual_node };
        vector<node> children = createChildren(actual_node);
        

        if (nodes.size()-1 + children.size() > num_threads) break;
        nodes.erase(nodes.begin());
        nodes.insert(nodes.end(), children.begin(), children.end());
    }
    printf("Se formaron %d nodos\n",nodes.size());
    return nodes;
}

struct state generatePreviousState(struct state actual_state, int movement) {
    int i_0 = actual_state.zero_index[0];
    int j_0 = actual_state.zero_index[1];

    switch (movement) {
    case 1:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0 + 1][j_0];
        actual_state.game_matrix[i_0 + 1][j_0] = 0;

        actual_state.zero_index[0] = i_0 + 1;
        break;
    case 2:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0][j_0 - 1];
        actual_state.game_matrix[i_0][j_0 - 1] = 0;

        actual_state.zero_index[1] = j_0 - 1;
        break;
    case 3:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0 - 1][j_0];
        actual_state.game_matrix[i_0 - 1][j_0] = 0;

        actual_state.zero_index[0] = i_0 - 1;
        break;
    case 4:
        actual_state.game_matrix[i_0][j_0] = actual_state.game_matrix[i_0][j_0 + 1];
        actual_state.game_matrix[i_0][j_0 + 1] = 0;

        actual_state.zero_index[1] = j_0 + 1;
        break;

    }

    return actual_state;
}

int main()
{
    clock_t start = clock();

    int initial_game_matrix[3][3] = {{ 0,3,5 },
                                     { 7,1,2 },
                                     { 4,8,6 }};

    /*int initial_game_matrix[3][3] = {{3,2,5}, //0.4 secs
                                     {6,0,8},
                                     {7,1,4}};*/

    /*int initial_game_matrix[3][3] = {{1,2,5},
                                     {3,4,8},
                                     {6,0,7}};*/

    struct state initial_state(initial_game_matrix);
    struct node initial_node(initial_state);

    vector<node> distributed_nodes = distributeThreads(total_threads, { initial_node });

    if (distributed_nodes.size() == 1 && validateState(distributed_nodes[0].node_state)) {
        printf("No se tuvo que hacer paralelo\n");
        //Se encontró la solución
    }
    else {
        printf("Se hace paralelo\n");
        printf("Nodos ejecutados %I64u\n\n", distributed_nodes.size());
        total_threads = distributed_nodes.size();

        vector<int> nodesHeigth;
        vector<__int8> memory;

        for (int i = 0; i < total_threads; i++) {
            nodesHeigth.push_back((int)distributed_nodes[i].secuence.size());
            struct state parent = generatePreviousState(distributed_nodes[i].node_state, distributed_nodes[i].secuence.back());

            for (int j = 0; j < 3; j++)
                for (int k = 0; k < 3; k++)
                    memory.push_back(parent.game_matrix[j][k]);
        }

        hipError_t cudaStatus = bfsWithCuda(&memory[0], &distributed_nodes[0], &nodesHeigth[0]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

    }
    clock_t end = clock();
    double elapsedTime = (double(end - start) / CLOCKS_PER_SEC);
    cout << "Tiempo de ejecucion: " << elapsedTime << endl;

    return 0;
}

// Función auxiliar para el lanzamiento del kernel
hipError_t bfsWithCuda(__int8* memory, struct node* nodes, int* heigths)
{
    __int8* dev_memory = 0;
    struct node* dev_nodes = 0;
    int* dev_heights = 0;
    
    hipError_t cudaStatus;

    // Elegir GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //Asignar espacio en la GPU
    cudaStatus = hipMalloc((void**)&dev_memory, total_threads * 9 * sizeof(__int8));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_nodes, total_threads * sizeof(struct node));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_heights, total_threads * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    //Copiar los valores de entrada en la GPU
    cudaStatus = hipMemcpy(dev_memory, memory, total_threads * 9 * sizeof(__int8), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_nodes, nodes, total_threads * sizeof(struct node), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_heights, heigths, total_threads * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //Llamar el kernel
    int threads_per_block = ceil(((double)total_threads) / ((double) blocks));
    printf("Numero de hilos por bloque: %d\n", threads_per_block);

    addKernel<<<blocks, threads_per_block >>>(dev_memory, dev_nodes, dev_heights);
    

    // Verificar errores de lanzamiento del kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // Espera a que termine el kernel y retorna algún error encontrado
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    //Free GPU memory
Error:
    hipFree(dev_memory);
    hipFree(dev_nodes);
    hipFree(dev_heights);
    
    return cudaStatus;
}